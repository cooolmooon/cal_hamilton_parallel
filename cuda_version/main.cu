#include "hip/hip_runtime.h"
#include<iostream>
#include"./tools/input.h"
#include"./tools/spline.h"
#include<sstream>
#include<cmath>
#include<stdio.h>
#include<iomanip>
#include"hip/hip_runtime.h"

using namespace std;
//-----------------------------------------------------------
//EXPLAIN:constant used by interplation
//-----------------------------------------------------------
__device__ double dr;
__device__ int cutoff; 
__device__ int width;
__device__ double fvalues[2000];
__device__ double m[2000];
__device__ int dev_point[2500]; 
__device__ float sum=0.0;
//-----------------------------------------------------------
//EXPLAIN:constant used by integral
//-----------------------------------------------------------
__device__ double tx;
__device__ double ty;
__device__ double tz;
__device__ int npoint;
__device__ int nx;
__device__ int ny;
__device__ int nz;
//-----------------------------------------------------------
//EXPLAIN:interplation
//-----------------------------------------------------------
__device__ double interplate(double r){
    int n=floor(r/dr);
    double p1=r/dr-n;
    double p2=n+1-r/dr;
    double ans=(1+2*p1)*p2*p2*fvalues[n]+(1+2*p2)*p1*p1*fvalues[n+1]
                +(r-n*dr)*p2*p2*m[n]+(r-(n+1)*dr)*p1*p1*m[n+1];
    return ans;
}
//-----------------------------------------------------------
//EXPLAIN:test
//-----------------------------------------------------------
__global__ void print_data(int* dev_area){
    printf("(%d,%d),(%d,%d),(%d,%d),\n",dev_point[0],dev_point[1],dev_point[2],dev_point[3],dev_point[4],dev_point[5]);
    printf("sum=%f\n",sum);
}
//-----------------------------------------------------------
//EXPLAIN:integral
//-----------------------------------------------------------
__global__ void integral(int* dev_area,int*dev_x,int* dev_y,int* dev_z,double* dev_V,float* dev_matH){
    //allocate dynamic SMEM
    // extern __shared__ int arr[];
    int k=blockIdx.x;
    int thdid=threadIdx.x;
    int pA=dev_point[2*k];
    int pB=dev_point[2*k+1];
    int sx=dev_area[npoint*pA+pB*6];
    int sy=dev_area[npoint*pA+pB*6+2];
    int sz=dev_area[npoint*pA+pB*6+4];
    int task_num=width*width/4;
    int start=thdid*task_num;
    // arr[0]=sx;
    // arr[1]=sy;
    // arr[2]=sz;
    // arr[3]=width;
    // arr[4]=dev_x[pA];
    // arr[5]=dev_y[pA];
    // arr[6]=dev_z[pA];
    // arr[7]=dev_x[pB];
    // arr[8]=dev_y[pB];
    // arr[9]=dev_z[pB];
    // arr[10]=ny;
    // arr[11]=nz;
    //printf("%d,%d,%d,%d,%d,%d\n",k,thdid,pA,pB,task_num,start);
    for(int i=0;i<task_num;i++){
        // int x=((start+i)/(arr[3]*arr[3])+arr[0]);
        // int y=(((start+i)%(arr[3]*arr[3]))/arr[3]+arr[1]);
        // int z=((start+i)%arr[3]+arr[2]);
        // //if(thdid==width)printf("block:%d, id:%d,%d, %d, %d\n",k,thdid,x,y,z);
        // double rA=sqrt((x*tx-arr[4])*(x*tx-arr[4])+
        // (y*ty-arr[5])*(y*ty-arr[5])+
        // (z*tz-arr[6])*(z*tz-arr[6]));
        // double rB=sqrt((x*tx-arr[7])*(x*tx-arr[7])+
        // (y*ty-arr[8])*(y*ty-arr[8])+
        // (z*tz-arr[9])*(z*tz-arr[9]));
        // double fA=interplate(rA);
        // double fB=interplate(rB);
        // float tmp=fA*fB*dev_V[x*arr[10]*arr[11]+y*arr[11]+z]*tx*ty*tz;
        int x=((start+i)/(width*width)+sx);
        int y=(((start+i)%(width*width))/width+sy);
        int z=((start+i)%width+sz);
        //if(thdid==width)printf("block:%d, id:%d,%d, %d, %d\n",k,thdid,x,y,z);
        double rA=sqrt((x*tx-dev_x[pA])*(x*tx-dev_x[pA])+
        (y*ty-dev_y[pA])*(y*ty-dev_y[pA])+
        (z*tz-dev_z[pA])*(z*tz-dev_z[pA]));
        double rB=sqrt((x*tx-dev_x[pB])*(x*tx-dev_x[pB])+
        (y*ty-dev_y[pB])*(y*ty-dev_y[pB])+
        (z*tz-dev_z[pB])*(z*tz-dev_z[pB]));
        double fA=interplate(rA);
        double fB=interplate(rB);
        float tmp=fA*fB*dev_V[x*ny*nz+y*nz+z]*tx*ty*tz;
        atomicAdd(dev_matH+pA*npoint+pB,tmp);
        __syncthreads();
    }
}
//-----------------------------------------------------------
//EXPLAIN:locate the start and end pos
//-----------------------------------------------------------
void compare(const int&x1,const int&x2,const int&cutoff,const int&lx,int&sx,int&ex,
            const int& width,const double& tx,const int&nx){
    sx=x1-cutoff;
    if(sx<0){
        sx=0;
        ex=width;
    }else{
        sx=(int)sx/tx;
        ex=sx+width;
        if(ex>nx-1){
            ex=nx-1;
            sx=ex-width;
        }
    }
}
int main()
{
//-----------------------------------------------------------
//EXPLAIN:read input
//-----------------------------------------------------------
    input in("./input/INPUT.txt");
    spline Spline(in.distribution_path);
//-----------------------------------------------------------
//EXPLAIN:copy static data to device
//-----------------------------------------------------------
    size_t bytes1=Spline.mesh*sizeof(double);
    hipMemcpyToSymbol(HIP_SYMBOL(dr),&Spline.dr,sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(cutoff),&Spline.cutoff,sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(fvalues),Spline.fvalues,bytes1);
    hipMemcpyToSymbol(HIP_SYMBOL(m),Spline.m,bytes1);
    double host_tx=(double)in.lx/in.nx;
    double host_ty=(double)in.ly/in.ny;
    double host_tz=(double)in.lz/in.nz;
    hipMemcpyToSymbol(HIP_SYMBOL(tx),&host_tx,sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(ty),&host_ty,sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(tz),&host_tz,sizeof(double));
    int host_width=2*Spline.cutoff/host_tx+2;
    if(host_width%2==1)host_width+=1;
    hipMemcpyToSymbol(HIP_SYMBOL(width),&host_width,sizeof(int));
//-----------------------------------------------------------
//EXPLAIN:preprocess
//-----------------------------------------------------------
    int* cal_area;
    int cal_point[2500];//2k stand for i, 2k+1 stand for j, k is blockIdx
    cal_area=new int[6*in.npoints*in.npoints];//the area that might intersect:160x160x160
    int cal_cnt=0;
    for(int i=0;i<6*in.npoints*in.npoints;i++){
        cal_area[i]=0;
    }
    for(int i=0;i<in.npoints;i++){
        int x1=in.pos_x[i];
        int y1=in.pos_y[i];
        int z1=in.pos_z[i];
        for(int j=i;j<in.npoints;j++){
            int x2=in.pos_x[j];
            int y2=in.pos_y[j];
            int z2=in.pos_z[j];
            if(abs(x1-x2)>Spline.cutoff||
            abs(y1-y2)>Spline.cutoff||
            abs(z1-z2)>Spline.cutoff){
                continue;
            }
            int sx,sy,sz,ex,ey,ez;
//-----------------------------------------------------------
//EXPLAIN:locate the start and end pos
//-----------------------------------------------------------
            compare(x1,x2,Spline.cutoff,in.lx,sx,ex,host_width,host_tx,in.nx);
            compare(y1,y2,Spline.cutoff,in.ly,sy,ey,host_width,host_tx,in.nx);
            compare(z1,z2,Spline.cutoff,in.lz,sz,ez,host_width,host_tx,in.nx);
            cal_area[in.npoints*i+6*j]=sx;
            cal_area[in.npoints*i+6*j+1]=ex;
            cal_area[in.npoints*i+6*j+2]=sy;
            cal_area[in.npoints*i+6*j+3]=ey;
            cal_area[in.npoints*i+6*j+4]=sz;
            cal_area[in.npoints*i+6*j+5]=ez;
            cal_point[cal_cnt*2]=i;
            cal_point[cal_cnt*2+1]=j;
            cal_cnt+=1;
        }
    }
    // for(int i=0;i<in.npoints;i++){
    //     for(int j=i;j<in.npoints;j++){
    //         for(int k=0;k<6;k++){
    //             std::cout<<cal_area[in.npoints*i+6*j+k]<<" ";
    //         }
    //         std::cout<<endl;
    //     }
    // }
    int* dev_area;
    size_t bytes2=6*in.npoints*in.npoints*sizeof(int);
    hipMalloc((int**)&dev_area,bytes2);
    hipMemcpy(dev_area,cal_area,bytes2,hipMemcpyHostToDevice);
    size_t bytes3=2000*sizeof(int);
    hipMemcpyToSymbol(HIP_SYMBOL(dev_point),cal_point,bytes3);
    hipMemcpyToSymbol(HIP_SYMBOL(npoint),&in.npoints,sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(nx),&in.nx,sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(ny),&in.ny,sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(nz),&in.nz,sizeof(int));
    int *dev_x;
    int *dev_y;
    int *dev_z;
    size_t bytes6=in.npoints*sizeof(int);
    hipMalloc((int**)&dev_x,bytes6);
    hipMalloc((int**)&dev_y,bytes6);
    hipMalloc((int**)&dev_z,bytes6);
    hipMemcpy(dev_x,in.pos_x,bytes6,hipMemcpyHostToDevice);
    hipMemcpy(dev_y,in.pos_y,bytes6,hipMemcpyHostToDevice);
    hipMemcpy(dev_z,in.pos_z,bytes6,hipMemcpyHostToDevice);
//-----------------------------------------------------------
//EXPLAIN:matrix H
//-----------------------------------------------------------
    float* host_matH;
    float* dev_matH;
    host_matH=new float[in.npoints*in.npoints];
    size_t bytes4=in.npoints*in.npoints*sizeof(float);
    hipMalloc((float**)&dev_matH,bytes4);
    double* dev_V;
    size_t bytes5=in.nx*in.ny*in.nz*sizeof(double);
    hipMalloc((double**)&dev_V,bytes5);
    hipMemcpy(dev_V,in.v_value,bytes5,hipMemcpyHostToDevice);
	//number of blocks
	int num_blk=cal_cnt;
	//number of threads
	int num_thd=host_width*4;
    // int num_thd=512;
    // print_data<<<1,1>>>(dev_area);
//-----------------------------------------------------------
//EXPLAIN:integral
//-----------------------------------------------------------
    hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0);
    integral<<<num_blk,num_thd>>>(dev_area,dev_x,dev_y,dev_z,dev_V,dev_matH);
    hipDeviceSynchronize();
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime,start,stop);
    std::cout<<"time used for integral is: "<<elapsedTime<<"ms"<<endl;
    // print_data<<<1,1>>>(dev_area);
//-----------------------------------------------------------
//EXPLAIN:copy result to host
//-----------------------------------------------------------
    hipMemcpy(host_matH,dev_matH,bytes4,hipMemcpyDeviceToHost);
    // float host_sum=2;
    // hipMemcpyFromSymbol(&host_sum,HIP_SYMBOL(sum),sizeof(float));
    // std::cout<<"sum= "<<host_sum<<endl;
    for(int i=0;i<in.npoints;i++){
        for(int j=0;j<i;j++){
            host_matH[i*in.npoints+j]=host_matH[j*in.npoints+i];
        }
    }
//-----------------------------------------------------------
//EXPLAIN:creater folder to save the output files
//-----------------------------------------------------------
    string fn="output";
    std::stringstream ss;
    ss<<"test -d "<<fn<<" || mkdir "<<fn;
    system(ss.str().c_str());
//-----------------------------------------------------------
//EXPLAIN:output result
//-----------------------------------------------------------
    ofstream ofs("./output/matrix.txt");
    ofs<<"dimension: "<<in.npoints<<endl;
    for(int i=0;i<in.npoints;i++){
        for(int j=0;j<in.npoints;j++){
            ofs<<setw(15)<<host_matH[i*in.npoints+j]<<" ";
        }
        ofs<<endl;
    }
    hipFree(dev_area);
    hipFree(dev_matH);
    hipFree(dev_V);
	hipDeviceReset();
    delete[] cal_area;
    delete[] host_matH;

    return 0;
}